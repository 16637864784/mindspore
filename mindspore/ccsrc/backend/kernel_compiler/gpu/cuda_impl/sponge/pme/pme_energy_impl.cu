#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "backend/kernel_compiler/gpu/cuda_impl/sponge/pme/pme_energy_impl.cuh"
#include "backend/kernel_compiler/gpu/cuda_impl/sponge/pme/pme_common.cuh"
#include "backend/kernel_compiler/gpu/cuda_impl/sponge/common_sponge.cuh"

__global__ void PME_Energy_Product(const int element_number, const float *list1, const float *list2, float *sum) {
  if (threadIdx.x == 0) {
    sum[0] = 0.;
  }
  __syncthreads();
  float lin = 0.0;
  for (int i = threadIdx.x; i < element_number; i = i + blockDim.x) {
    lin = lin + list1[i] * list2[i];
  }
  atomicAdd(sum, lin);
}

__global__ void PME_Energy_Reciprocal(const int element_number, const hipfftComplex *FQ, const float *BC, float *sum) {
  if (threadIdx.x == 0) {
    sum[0] = 0.;
  }
  __syncthreads();
  float lin = 0.0;
  hipfftComplex FQ_i;
  for (int i = threadIdx.x; i < element_number; i = i + blockDim.x) {
    FQ_i = FQ[i];
    lin = lin + (FQ_i.x * FQ_i.x + FQ_i.y * FQ_i.y) * BC[i];
  }
  atomicAdd(sum, lin);
}

__global__ void PME_Excluded_Energy_Correction(const int atom_numbers, const UNSIGNED_INT_VECTOR *uint_crd,
                                               const VECTOR *sacler, const float *charge, const float pme_beta,
                                               const float sqrt_pi, const int *excluded_list_start,
                                               const int *excluded_list, const int *excluded_atom_numbers, float *ene) {
  int atom_i = blockDim.x * blockIdx.x + threadIdx.x;
  if (atom_i < atom_numbers) {
    int excluded_number = excluded_atom_numbers[atom_i];
    if (excluded_number > 0) {
      int list_start = excluded_list_start[atom_i];
      // int atom_min = excluded_list[list_start];
      int list_end = list_start + excluded_number;
      int atom_j;
      int int_x;
      int int_y;
      int int_z;

      float charge_i = charge[atom_i];
      float charge_j;
      float dr_abs;
      float beta_dr;

      UNSIGNED_INT_VECTOR r1 = uint_crd[atom_i], r2;
      VECTOR dr;
      float dr2;

      float ene_lin = 0.;

      for (int i = list_start; i < list_end; i = i + 1) {
        atom_j = excluded_list[i];
        r2 = uint_crd[atom_j];
        charge_j = charge[atom_j];

        int_x = r2.uint_x - r1.uint_x;
        int_y = r2.uint_y - r1.uint_y;
        int_z = r2.uint_z - r1.uint_z;
        dr.x = sacler[0].x * int_x;
        dr.y = sacler[0].y * int_y;
        dr.z = sacler[0].z * int_z;
        dr2 = dr.x * dr.x + dr.y * dr.y + dr.z * dr.z;

        dr_abs = sqrtf(dr2);
        beta_dr = pme_beta * dr_abs;

        ene_lin -= charge_i * charge_j * erff(beta_dr) / dr_abs;
      }
      atomicAdd(ene, ene_lin);
    }
  }
}

void PMEEnergy(int fftx, int ffty, int fftz, int atom_numbers, float beta, float *box_length_f, float *PME_BC,
               int *pme_uxyz, float *pme_frxyz, float *PME_Q, float *pme_fq, int *PME_atom_near, int *pme_kxyz,
               const int *uint_crd_f, const float *charge, int *nl_atom_numbers, int *nl_atom_serial, int *nl,
               const float *scaler_f, const int *excluded_list_start, const int *excluded_list,
               const int *excluded_atom_numbers, float *d_reciprocal_ene, float *d_self_ene, float *d_direct_ene,
               float *d_correction_ene, hipStream_t stream) {
  UNSIGNED_INT_VECTOR *uint_crd =
    const_cast<UNSIGNED_INT_VECTOR *>(reinterpret_cast<const UNSIGNED_INT_VECTOR *>(uint_crd_f));
  VECTOR *scaler = const_cast<VECTOR *>(reinterpret_cast<const VECTOR *>(scaler_f));
  int max_neighbor_numbers = 800;
  NEIGHBOR_LIST *nl_a = reinterpret_cast<NEIGHBOR_LIST *>(nl);
  construct_neighbor_list_kernel<<<ceilf(static_cast<float>(atom_numbers) / 128), 128, 0, stream>>>(
    atom_numbers, max_neighbor_numbers, nl_atom_numbers, nl_atom_serial, nl_a);
  std::vector<float> h_box_length(3);
  hipMemcpyAsync(h_box_length.data(), box_length_f, sizeof(float) * h_box_length.size(), hipMemcpyDeviceToHost,
                  stream);
  hipStreamSynchronize(stream);
  VECTOR *box_length = reinterpret_cast<VECTOR *>(h_box_length.data());

  UNSIGNED_INT_VECTOR *PME_uxyz = reinterpret_cast<UNSIGNED_INT_VECTOR *>(pme_uxyz);
  UNSIGNED_INT_VECTOR *PME_kxyz = reinterpret_cast<UNSIGNED_INT_VECTOR *>(pme_kxyz);
  VECTOR *PME_frxyz = reinterpret_cast<VECTOR *>(pme_frxyz);
  hipfftComplex *PME_FQ = reinterpret_cast<hipfftComplex *>(pme_fq);
  hipfftHandle PME_plan_r2c;
  hipfftHandle PME_plan_c2r;
  hipfftPlan3d(&PME_plan_r2c, fftx, ffty, fftz, HIPFFT_R2C);
  hipfftPlan3d(&PME_plan_c2r, fftx, ffty, fftz, HIPFFT_C2R);
  hipfftSetStream(PME_plan_r2c, stream);
  hipfftSetStream(PME_plan_c2r, stream);
  thread_PME.x = 8;
  thread_PME.y = 8;
  int PME_Nin = ffty * fftz;
  int PME_Nfft = fftx * ffty * (fftz / 2 + 1);
  int PME_Nall = fftx * ffty * fftz;
  float volume = box_length[0].x * box_length[0].y * box_length[0].z;

  UNSIGNED_INT_VECTOR *PME_kxyz_cpu;
  Malloc_Safely(reinterpret_cast<void **>(&PME_kxyz_cpu), sizeof(UNSIGNED_INT_VECTOR) * 64);

  int kx, ky, kz, kxrp, kyrp, kzrp, index;
  for (kx = 0; kx < 4; kx++) {
    for (ky = 0; ky < 4; ky++) {
      for (kz = 0; kz < 4; kz++) {
        index = kx * 16 + ky * 4 + kz;
        PME_kxyz_cpu[index].uint_x = kx;
        PME_kxyz_cpu[index].uint_y = ky;
        PME_kxyz_cpu[index].uint_z = kz;
      }
    }
  }
  hipMemcpyAsync(PME_kxyz, PME_kxyz_cpu, sizeof(UNSIGNED_INT_VECTOR) * 64, hipMemcpyHostToDevice, stream);
  hipStreamSynchronize(stream);
  free(PME_kxyz_cpu);

  // initial start
  float *B1, *B2, *B3, *PME_BC0;
  B1 = reinterpret_cast<float *>(malloc(sizeof(float) * fftx));
  B2 = reinterpret_cast<float *>(malloc(sizeof(float) * ffty));
  B3 = reinterpret_cast<float *>(malloc(sizeof(float) * fftz));
  PME_BC0 = reinterpret_cast<float *>(malloc(sizeof(float) * PME_Nfft));

  for (kx = 0; kx < fftx; kx++) {
    B1[kx] = getb(kx, fftx, 4);
  }

  for (ky = 0; ky < ffty; ky++) {
    B2[ky] = getb(ky, ffty, 4);
  }

  for (kz = 0; kz < fftz; kz++) {
    B3[kz] = getb(kz, fftz, 4);
  }
  float mprefactor = PI * PI / -beta / beta;

  float msq;
  for (kx = 0; kx < fftx; kx++) {
    kxrp = kx;
    if (kx > fftx / 2) kxrp = fftx - kx;
    for (ky = 0; ky < ffty; ky++) {
      kyrp = ky;
      if (ky > ffty / 2) kyrp = ffty - ky;
      for (kz = 0; kz <= fftz / 2; kz++) {
        kzrp = kz;

        msq = kxrp * kxrp / box_length[0].x / box_length[0].x + kyrp * kyrp / box_length[0].y / box_length[0].y +
              kzrp * kzrp / box_length[0].z / box_length[0].z;
        index = kx * ffty * (fftz / 2 + 1) + ky * (fftz / 2 + 1) + kz;
        if ((kx + ky + kz) == 0) {
          PME_BC0[index] = 0;
        } else {
          PME_BC0[index] = 1.0 / PI / msq * exp(mprefactor * msq) / volume;
        }

        PME_BC0[index] *= B1[kx] * B2[ky] * B3[kz];
      }
    }
  }

  hipMemcpyAsync(PME_BC, PME_BC0, sizeof(float) * PME_Nfft, hipMemcpyHostToDevice, stream);
  hipStreamSynchronize(stream);
  free(B1);
  free(B2);
  free(B3);
  free(PME_BC0);

  Reset_List<<<3 * atom_numbers / 32 + 1, 32, 0, stream>>>(3 * atom_numbers, reinterpret_cast<int *>(PME_uxyz),
                                                           1 << 30);
  PME_Atom_Near<<<atom_numbers / 32 + 1, 32, 0, stream>>>(
    uint_crd, PME_atom_near, PME_Nin, periodic_factor_inverse * fftx, periodic_factor_inverse * ffty,
    periodic_factor_inverse * fftz, atom_numbers, fftx, ffty, fftz, PME_kxyz, PME_uxyz, PME_frxyz);

  Reset_List<<<PME_Nall / 1024 + 1, 1024, 0, stream>>>(PME_Nall, PME_Q, 0);

  PME_Q_Spread<<<atom_numbers / thread_PME.x + 1, thread_PME, 0, stream>>>(PME_atom_near, charge, PME_frxyz, PME_Q,
                                                                           PME_kxyz, atom_numbers);

  hipfftExecR2C(PME_plan_r2c, reinterpret_cast<float *>(PME_Q), reinterpret_cast<hipfftComplex *>(PME_FQ));

  PME_Energy_Reciprocal<<<1, 1024, 0, stream>>>(PME_Nfft, PME_FQ, PME_BC, d_reciprocal_ene);

  PME_Energy_Product<<<1, 1024, 0, stream>>>(atom_numbers, charge, charge, d_self_ene);
  Scale_List<<<1, 1, 0, stream>>>(1, d_self_ene, -beta / sqrtf(PI));

  Reset_List<<<1, 1, 0, stream>>>(1, d_direct_ene, 0.0);
  PME_Direct_Energy<<<atom_numbers / thread_PME.x + 1, thread_PME, 0, stream>>>(
    atom_numbers, nl_a, uint_crd, scaler, charge, beta, cutoff * cutoff, d_direct_ene);

  Reset_List<<<1, 1, 0, stream>>>(1, d_correction_ene, 0.0);
  PME_Excluded_Energy_Correction<<<atom_numbers / 32 + 1, 32, 0, stream>>>(
    atom_numbers, uint_crd, scaler, charge, beta, sqrtf(PI), excluded_list_start, excluded_list, excluded_atom_numbers,
    d_correction_ene);
  return;
}
void PMEEnergy(int fftx, int ffty, int fftz, int atom_numbers, float beta, float *box_length_f, float *PME_BC,
               int *pme_uxyz, float *pme_frxyz, float *PME_Q, float *pme_fq, int *PME_atom_near, int *pme_kxyz,
               const int *uint_crd_f, const float *charge, int *nl_atom_numbers, int *nl_atom_serial, int *nl,
               const float *scaler_f, const int *excluded_list_start, const int *excluded_list,
               const int *excluded_atom_numbers, float *d_reciprocal_ene, float *d_self_ene, float *d_direct_ene,
               float *d_correction_ene, hipStream_t stream);
