#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "backend/kernel_compiler/gpu/cuda_impl/sponge/vatom/v3_coordinate_refresh_impl.cuh"
#include "backend/kernel_compiler/gpu/cuda_impl/sponge/common_sponge.cuh"

__global__ void v3_Coordinate_Refresh(const int virtual_numbers, const VIRTUAL_TYPE_3 *v_info,
                                      const UNSIGNED_INT_VECTOR *uint_crd, const VECTOR scaler, VECTOR *coordinate) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < virtual_numbers) {
    VIRTUAL_TYPE_3 v_temp = v_info[i];
    int atom_v = v_temp.virtual_atom;
    int atom_1 = v_temp.from_1;
    int atom_2 = v_temp.from_2;
    int atom_3 = v_temp.from_3;
    float d = v_temp.d;
    float k = v_temp.k;
    UNSIGNED_INT_VECTOR uint_r1 = uint_crd[atom_1];
    UNSIGNED_INT_VECTOR uint_r2 = uint_crd[atom_2];
    UNSIGNED_INT_VECTOR uint_r3 = uint_crd[atom_3];

    VECTOR r21 = Get_Periodic_Displacement(uint_r2, uint_r1, scaler);
    VECTOR r32 = Get_Periodic_Displacement(uint_r3, uint_r2, scaler);

    VECTOR temp = r21 + k * r32;
    temp = d * rnorm3df(temp.x, temp.y, temp.z) * temp;
    coordinate[atom_v] = coordinate[atom_1] + temp;
  }
}

void v3CoordinateRefresh(int atom_numbers, int virtual_numbers, const float *v_info_f, const int *uint_crd_f,
                         const float *scaler_f, float *crd_f, hipStream_t stream) {
  Reset_List<<<ceilf(static_cast<float>(3. * atom_numbers) / 128), 128, 0, stream>>>(3 * atom_numbers, crd_f, 0.);
  size_t thread_per_block = 128;
  size_t block_per_grid = ceilf(static_cast<float>(virtual_numbers) / 128);
  UNSIGNED_INT_VECTOR *uint_crd =
    const_cast<UNSIGNED_INT_VECTOR *>(reinterpret_cast<const UNSIGNED_INT_VECTOR *>(uint_crd_f));

  VECTOR *scaler = const_cast<VECTOR *>(reinterpret_cast<const VECTOR *>(scaler_f));
  VECTOR *crd = const_cast<VECTOR *>(reinterpret_cast<const VECTOR *>(crd_f));
  VIRTUAL_TYPE_3 *v_info = const_cast<VIRTUAL_TYPE_3 *>(reinterpret_cast<const VIRTUAL_TYPE_3 *>(v_info_f));

  v3_Coordinate_Refresh<<<block_per_grid, thread_per_block, 0, stream>>>(virtual_numbers, v_info, uint_crd, scaler[0],
                                                                         crd);

  return;
}

void v3CoordinateRefresh(int atom_numbers, int virtual_numbers, const float *v_info_f, const int *uint_crd_f,
                         const float *scaler_f, float *crd_f, hipStream_t stream);
