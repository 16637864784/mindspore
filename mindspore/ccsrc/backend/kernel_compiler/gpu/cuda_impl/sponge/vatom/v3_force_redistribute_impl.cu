#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "backend/kernel_compiler/gpu/cuda_impl/sponge/vatom/v3_force_redistribute_impl.cuh"
#include "backend/kernel_compiler/gpu/cuda_impl/sponge/common_sponge.cuh"
#include "backend/kernel_compiler/gpu/cuda_impl/util.cuh"

__global__ void v3_Force_Redistribute(const int virtual_numbers, const VIRTUAL_TYPE_3 *v_info,
                                      const UNSIGNED_INT_VECTOR *uint_crd, const VECTOR scaler, VECTOR *force) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < virtual_numbers) {
    VIRTUAL_TYPE_3 v_temp = v_info[i];
    int atom_v = v_temp.virtual_atom;
    int atom_1 = v_temp.from_1;
    int atom_2 = v_temp.from_2;
    int atom_3 = v_temp.from_3;
    float d = v_temp.d;
    float k = v_temp.k;
    VECTOR force_v = force[atom_v];

    UNSIGNED_INT_VECTOR uint_r1 = uint_crd[atom_1];
    UNSIGNED_INT_VECTOR uint_r2 = uint_crd[atom_2];
    UNSIGNED_INT_VECTOR uint_r3 = uint_crd[atom_3];
    UNSIGNED_INT_VECTOR uint_rv = uint_crd[atom_v];

    VECTOR r21 = Get_Periodic_Displacement(uint_r2, uint_r1, scaler);
    VECTOR r32 = Get_Periodic_Displacement(uint_r3, uint_r2, scaler);
    VECTOR rv1 = Get_Periodic_Displacement(uint_rv, uint_r1, scaler);

    VECTOR temp = r21 + k * r32;
    float factor = d * rnorm3df(temp.x, temp.y, temp.z);

    temp = (rv1 * force_v) / (rv1 * rv1) * rv1;
    temp = factor * (force_v - temp);
    VECTOR force_1 = force_v - temp;
    VECTOR force_2 = (1 - k) * temp;
    VECTOR force_3 = k * temp;

    atomicAdd(&force[atom_1].x, force_1.x);
    atomicAdd(&force[atom_1].y, force_1.y);
    atomicAdd(&force[atom_1].z, force_1.z);

    atomicAdd(&force[atom_2].x, force_2.x);
    atomicAdd(&force[atom_2].y, force_2.y);
    atomicAdd(&force[atom_2].z, force_2.z);

    atomicAdd(&force[atom_3].x, force_3.x);
    atomicAdd(&force[atom_3].y, force_3.y);
    atomicAdd(&force[atom_3].z, force_3.z);

    force_v.x = 0.0f;
    force_v.y = 0.0f;
    force_v.z = 0.0f;
    force[atom_v] = force_v;
  }
}

void v3ForceRedistribute(int atom_numbers, int virtual_numbers, const float *v_info_f, const int *uint_crd_f,
                         const float *scaler_f, float *frc_f, hipStream_t stream) {
  Reset_List<<<ceilf(static_cast<float>(3. * atom_numbers) / 128), 128, 0, stream>>>(3 * atom_numbers, frc_f, 0.);
  size_t thread_per_block = 128;
  size_t block_per_grid = ceilf(static_cast<float>(virtual_numbers) / 128);
  UNSIGNED_INT_VECTOR *uint_crd =
    const_cast<UNSIGNED_INT_VECTOR *>(reinterpret_cast<const UNSIGNED_INT_VECTOR *>(uint_crd_f));

  VECTOR *scaler = const_cast<VECTOR *>(reinterpret_cast<const VECTOR *>(scaler_f));
  VECTOR *frc = const_cast<VECTOR *>(reinterpret_cast<const VECTOR *>(frc_f));
  VIRTUAL_TYPE_3 *v_info = const_cast<VIRTUAL_TYPE_3 *>(reinterpret_cast<const VIRTUAL_TYPE_3 *>(v_info_f));

  v3_Force_Redistribute<<<block_per_grid, thread_per_block, 0, stream>>>(virtual_numbers, v_info, uint_crd, scaler[0],
                                                                         frc);

  return;
}

void v3ForceRedistribute(int atom_numbers, int virtual_numbers, const float *v_info_f, const int *uint_crd_f,
                         const float *scaler_f, float *frc_f, hipStream_t stream);
