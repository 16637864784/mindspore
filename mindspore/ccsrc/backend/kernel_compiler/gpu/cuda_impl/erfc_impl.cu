#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */ 

#include "erfc_impl.cuh"
template <typename T>
__global__ void ErfcKernel(T *input, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = (T)erfc(input[i]);
  }
  return;
}

template <typename T>
void Erfc(T *input, T *output, size_t count, hipStream_t hip_stream) {
  ErfcKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, output, count);
  return;
}

template void Erfc<float>(float *input, float *output, size_t count, hipStream_t hip_stream);
