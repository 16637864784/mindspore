#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "backend/kernel_compiler/gpu/cuda_impl/relu_impl.cuh"
#include "runtime/device/gpu/cuda_common.h"

template <typename T>
__global__ void CalReLUKernel(int size, T *input_addr, T *output_addr) {
  for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    output_addr[pos] = input_addr[pos] > static_cast<T>(0) ? input_addr[pos] : static_cast<T>(0);
  }
}

template <typename T>
void CalReLU(int size, T *input_addr, T *output_addr, hipStream_t hip_stream) {
  CalReLUKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input_addr, output_addr);
  return;
}

template void CalReLU(int size, float *input_addr, float *output_addr, hipStream_t hip_stream);
template void CalReLU(int size, half *input_addr, half *output_addr, hipStream_t hip_stream);
template void CalReLU(int size, int32_t *input_addr, int32_t *output_addr, hipStream_t hip_stream);
template void CalReLU(int size, int64_t *input_addr, int64_t *output_addr, hipStream_t hip_stream);
