#include "hip/hip_runtime.h"
/**
 * Copyright 2019-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <algorithm>
#include "backend/kernel_compiler/gpu/cuda_impl/slice_impl.cuh"

// for each dimension, an explicit instantiation is used to generate this function
// during compile time, and the inner loop of all the generated functions should be
// unrolled by the compiler
template <typename T, typename...S>
__global__ void Slice(const T *input, T *output, const size_t output_size, S...pack) {
  const int unpacked[] = { pack... };
  const int param_list_size = static_cast<int>((sizeof...(pack)) / 3);
  const int slice_start_start = 0;
  const int slice_size_start = param_list_size;
  const int input_shape_start = 2 * param_list_size;

  for (size_t gt_id = blockIdx.x * blockDim.x + threadIdx.x; gt_id < output_size; gt_id += blockDim.x * gridDim.x) {
    int linear_index = gt_id;
    int output_stride = 1;
    int input_stride = 1;
    int input_offset = 0;

    for (int i = 0; i < param_list_size; i++) {
      int unravel_dimension = unpacked[slice_size_start + param_list_size - 1 - i];
      int unraveled_index = (linear_index / output_stride) % unravel_dimension;
      input_offset += (unraveled_index + unpacked[slice_start_start + param_list_size - 1 - i]) * input_stride;
      output_stride *= unravel_dimension;
      input_stride *= unpacked[input_shape_start + param_list_size - 1 - i];
    }

    output[gt_id] = input[input_offset];
  }
}

template <typename T, typename...S>
void SliceKernel(const T *input, T *output, const size_t output_size, hipStream_t hip_stream, S...pack) {
  Slice<<<GET_BLOCKS(sizeof...(pack)), GET_THREADS, 0, hip_stream>>>(input, output, output_size, pack...);
}

template <typename T>
__global__ void Slice4DGrad(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                        const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                        const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                        const T *dy, T *dx) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (l1 * l2 * l3 * l4); pos += blockDim.x * gridDim.x) {
    size_t i = pos / (l2 * l3 * l4) % l1;
    size_t j = pos / (l3 * l4) % l2;
    size_t k = pos / l4 % l3;
    size_t o = pos % l4;
    size_t input_idx = (i + s1) * (d2 * d3 * d4) + (j + s2) * (d3 * d4) + (k + s3) * d4 + (o + s4);
    dx[input_idx] = dy[pos];
  }
}

template <typename T>
__global__ void FillArray(T *addr, const size_t len, const float value) {
  T value_ = static_cast<T>(value);
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < len; pos += blockDim.x * gridDim.x) {
    addr[pos] = value_;
  }
  return;
}
template <typename T>
void FillDeviceArray(const size_t input_size, T *addr, const float value, hipStream_t hip_stream) {
  FillArray<<<GET_BLOCKS(input_size), GET_THREADS, 0, hip_stream>>>(addr, input_size, value);
  return;
}

template <typename T>
void CalSlice4DGrad(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t l1,
                   const size_t l2, const size_t l3, const size_t l4, const size_t d1, const size_t d2,
                   const size_t d3, const size_t d4, const T *dy, T *dx, hipStream_t stream) {
  Slice4DGrad<<<GET_BLOCKS(l1 * l2 * l3 * l4), GET_THREADS, 0, stream>>>(s1, s2, s3, s4, l1, l2, l3, l4, d1, d2, d3, d4,
                                                                     dy, dx);
}

template <typename T>
__global__ void StridedSliceKernel(const size_t b0, const size_t b1, const size_t b2, const size_t b3, const size_t b4,
                                   const size_t b5, const size_t b6, const size_t s0, const size_t s1, const size_t s2,
                                   const size_t s3, const size_t s4, const size_t s5, const size_t s6, const size_t i0,
                                   const size_t i1, const size_t i2, const size_t i3, const size_t i4, const size_t i5,
                                   const size_t i6, const size_t o0, const size_t o1, const size_t o2, const size_t o3,
                                   const size_t o4, const size_t o5, const size_t o6, const T *input_addr,
                                   T *output_addr) {
  size_t output_num = o0 * o1 * o2 * o3 * o4 * o5 * o6;
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < output_num; pos += blockDim.x * gridDim.x) {
    size_t i = pos / (o1 * o2 * o3 * o4 * o5 * o6) % o0;
    size_t j = pos / (o2 * o3 * o4 * o5 * o6) % o1;
    size_t k = pos / (o3 * o4 * o5 * o6) % o2;
    size_t l = pos / (o4 * o5 * o6) % o3;
    size_t m = pos / (o5 * o6) % o4;
    size_t n = pos / (o6) % o5;
    size_t o = pos % o6;

    size_t input_idx = (i * s0 + b0) * i1 * i2 * i3 * i4 * i5 * i6 + (j * s1 + b1) * i2 * i3 * i4 * i5 * i6 +
                       (k * s2 + b2) * i3 * i4 * i5 * i6 + (l * s3 + b3) * i4 * i5 * i6 + (m * s4 + b4) * i5 * i6 +
                       (n * s5 + b5) * i6 + (o * s6 + b6);
    output_addr[pos] = input_addr[input_idx];
  }
}

template <typename T>
void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                  const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape, const T *input,
                  T *output, hipStream_t hip_stream) {
  size_t size = output_shape[0] * output_shape[1] * output_shape[2] * output_shape[3] * output_shape[4] *
                output_shape[5] * output_shape[6];
  StridedSliceKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(
    begin[0], begin[1], begin[2], begin[3], begin[4], begin[5], begin[6], strides[0], strides[1], strides[2],
    strides[3], strides[4], strides[5], strides[6], input_shape[0], input_shape[1], input_shape[2], input_shape[3],
    input_shape[4], input_shape[5], input_shape[6], output_shape[0], output_shape[1], output_shape[2], output_shape[3],
    output_shape[4], output_shape[5], output_shape[6], input, output);
}

template <typename T>
__global__ void StridedSliceGradKernel(const size_t b0, const size_t b1, const size_t b2, const size_t b3,
                                       const size_t b4, const size_t b5, const size_t b6, const size_t s0,
                                       const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                       const size_t s5, const size_t s6, const size_t i0, const size_t i1,
                                       const size_t i2, const size_t i3, const size_t i4, const size_t i5,
                                       const size_t i6, const size_t o0, const size_t o1, const size_t o2,
                                       const size_t o3, const size_t o4, const size_t o5, const size_t o6, const T *dy,
                                       T *dx) {
  size_t output_num = o0 * o1 * o2 * o3 * o4 * o5 * o6;
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < output_num; pos += blockDim.x * gridDim.x) {
    size_t i = pos / (o1 * o2 * o3 * o4 * o5 * o6) % o0;
    size_t j = pos / (o2 * o3 * o4 * o5 * o6) % o1;
    size_t k = pos / (o3 * o4 * o5 * o6) % o2;
    size_t l = pos / (o4 * o5 * o6) % o3;
    size_t m = pos / (o5 * o6) % o4;
    size_t n = pos / (o6) % o5;
    size_t o = pos % o6;

    size_t input_idx = (i * s0 + b0) * i1 * i2 * i3 * i4 * i5 * i6 + (j * s1 + b1) * i2 * i3 * i4 * i5 * i6 +
                       (k * s2 + b2) * i3 * i4 * i5 * i6 + (l * s3 + b3) * i4 * i5 * i6 + (m * s4 + b4) * i5 * i6 +
                       (n * s5 + b5) * i6 + (o * s6 + b6);
    dx[input_idx] = dy[pos];
  }
  return;
}

template <typename T>
void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                      const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape, const T *dy, T *dx,
                      hipStream_t hip_stream) {
  size_t size = dy_shape[0] * dy_shape[1] * dy_shape[2] * dy_shape[3] * dy_shape[4] * dy_shape[5] * dy_shape[6];
  StridedSliceGradKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(
    begin[0], begin[1], begin[2], begin[3], begin[4], begin[5], begin[6], strides[0], strides[1], strides[2],
    strides[3], strides[4], strides[5], strides[6], dx_shape[0], dx_shape[1], dx_shape[2], dx_shape[3], dx_shape[4],
    dx_shape[5], dx_shape[6], dy_shape[0], dy_shape[1], dy_shape[2], dy_shape[3], dy_shape[4], dy_shape[5], dy_shape[6],
    dy, dx);
}

template void CalSlice4DGrad<double>(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                     const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                     const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                     const double *dy, double *dx, hipStream_t stream);
template void CalSlice4DGrad<float>(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t l1,
                                    const size_t l2, const size_t l3, const size_t l4, const size_t d1, const size_t d2,
                                    const size_t d3, const size_t d4, const float *dy, float *dx, hipStream_t stream);
template void CalSlice4DGrad<half>(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t l1,
                                   const size_t l2, const size_t l3, const size_t l4, const size_t d1, const size_t d2,
                                   const size_t d3, const size_t d4, const half *dy, half *dx, hipStream_t stream);
template void CalSlice4DGrad<int>(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t l1,
                                  const size_t l2, const size_t l3, const size_t l4, const size_t d1, const size_t d2,
                                  const size_t d3, const size_t d4, const int *dy, int *dx, hipStream_t stream);
template void CalSlice4DGrad<short>(const size_t s1, const size_t s2, const size_t s3, const size_t s4,  // NOLINT
                                    const size_t l1,
                                    const size_t l2, const size_t l3, const size_t l4, const size_t d1, const size_t d2,
                                    const size_t d3, const size_t d4, const short *dy, short *dx,  // NOLINT
                                    hipStream_t stream);
template void CalSlice4DGrad<unsigned char>(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const unsigned char *dy, unsigned char *dx, hipStream_t stream);
template void CalSlice4DGrad<int64_t>(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                      const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                      const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                      const int64_t *dy, int64_t *dx, hipStream_t stream);
template void CalSlice4DGrad<bool>(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t l1,
                                   const size_t l2, const size_t l3, const size_t l4, const size_t d1, const size_t d2,
                                   const size_t d3, const size_t d4, const bool *dy, bool *dx, hipStream_t stream);

template void FillDeviceArray<bool>(const size_t input_size, bool *addr, const float value, hipStream_t hip_stream);
template void FillDeviceArray<int64_t>(const size_t input_size, int64_t *addr, const float value,
                                       hipStream_t hip_stream);
template void FillDeviceArray<int>(const size_t input_size, int *addr, const float value, hipStream_t hip_stream);
template void FillDeviceArray<short>(const size_t input_size, short *addr, const float value,  // NOLINT
                                     hipStream_t hip_stream);
template void FillDeviceArray<int8_t>(const size_t input_size, int8_t *addr, const float value,
                                      hipStream_t hip_stream);
template void FillDeviceArray<uint64_t>(const size_t input_size, uint64_t *addr, const float value,
                                        hipStream_t hip_stream);
template void FillDeviceArray<uint32_t>(const size_t input_size, uint32_t *addr, const float value,
                                        hipStream_t hip_stream);
template void FillDeviceArray<uint16_t>(const size_t input_size, uint16_t *addr, const float value,
                                        hipStream_t hip_stream);
template void FillDeviceArray<unsigned char>(const size_t input_size, unsigned char *addr, const float value,
                                             hipStream_t hip_stream);
template void FillDeviceArray<half>(const size_t input_size, half *addr, const float value, hipStream_t hip_stream);
template void FillDeviceArray<float>(const size_t input_size, float *addr, const float value, hipStream_t hip_stream);
template void FillDeviceArray<double>(const size_t input_size, double *addr, const float value,
                                      hipStream_t hip_stream);

template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                           const bool *input, bool *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                           const double *input, double *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                           const float *input, float *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                           const half *input, half *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                           const int64_t *input, int64_t *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                           const int *input, int *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                           const short *input, short *output, hipStream_t hip_stream);  // NOLINT
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                           const int8_t *input, int8_t *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                           const uint64_t *input, uint64_t *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                           const uint32_t *input, uint32_t *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                           const uint16_t *input, uint16_t *output, hipStream_t hip_stream);
template void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                           const unsigned char *input, unsigned char *output, hipStream_t hip_stream);

template void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape, const bool *dy,
                               bool *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                               const double *dy, double *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                               const float *dy, float *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape, const half *dy,
                               half *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                               const int64_t *dy, int64_t *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape, const int *dy,
                               int *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                               const short *dy,                       // NOLINT
                               short *dx, hipStream_t hip_stream);  // NOLINT
template void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                               const int8_t *dy, int8_t *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                               const uint64_t *dy, uint64_t *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                               const uint32_t *dy, uint32_t *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                               const uint16_t *dy, uint16_t *dx, hipStream_t hip_stream);
template void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                               const unsigned char *dy, unsigned char *dx, hipStream_t hip_stream);

// add additional explicit instantiations here for additional dimensions
// bool
template void SliceKernel(const bool *input, bool *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t);

template void SliceKernel(const bool *input, bool *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const bool *input, bool *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const bool *input, bool *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t);

template void SliceKernel(const bool *input, bool *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const bool *input, bool *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const bool *input, bool *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

// uchar
template void SliceKernel(const unsigned char *input, unsigned char *output, const size_t output_size,
                          hipStream_t stream, int32_t, int32_t, int32_t);

template void SliceKernel(const unsigned char *input, unsigned char *output, const size_t output_size,
                          hipStream_t stream, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const unsigned char *input, unsigned char *output, const size_t output_size,
                          hipStream_t stream, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t);

template void SliceKernel(const unsigned char *input, unsigned char *output, const size_t output_size,
                          hipStream_t stream, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const unsigned char *input, unsigned char *output, const size_t output_size,
                          hipStream_t stream, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const unsigned char *input, unsigned char *output, const size_t output_size,
                          hipStream_t stream, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const unsigned char *input, unsigned char *output, const size_t output_size,
                          hipStream_t stream, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t);

// int16_t
template void SliceKernel(const int16_t *input, int16_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t);

template void SliceKernel(const int16_t *input, int16_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const int16_t *input, int16_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const int16_t *input, int16_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t);

template void SliceKernel(const int16_t *input, int16_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const int16_t *input, int16_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const int16_t *input, int16_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

// int32_t
template void SliceKernel(const int32_t *input, int32_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t);

template void SliceKernel(const int32_t *input, int32_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const int32_t *input, int32_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const int32_t *input, int32_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t);

template void SliceKernel(const int32_t *input, int32_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const int32_t *input, int32_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const int32_t *input, int32_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

// int64_t
template void SliceKernel(const int64_t *input, int64_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t);

template void SliceKernel(const int64_t *input, int64_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const int64_t *input, int64_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const int64_t *input, int64_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t);

template void SliceKernel(const int64_t *input, int64_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const int64_t *input, int64_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const int64_t *input, int64_t *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

// half
template void SliceKernel(const half *input, half *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t);

template void SliceKernel(const half *input, half *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const half *input, half *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const half *input, half *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t);

template void SliceKernel(const half *input, half *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const half *input, half *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const half *input, half *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

// float
template void SliceKernel(const float *input, float *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t);

template void SliceKernel(const float *input, float *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const float *input, float *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const float *input, float *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t);

template void SliceKernel(const float *input, float *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const float *input, float *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const float *input, float *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

// double
template void SliceKernel(const double *input, double *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t);

template void SliceKernel(const double *input, double *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const double *input, double *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const double *input, double *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t);

template void SliceKernel(const double *input, double *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const double *input, double *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);

template void SliceKernel(const double *input, double *output, const size_t output_size, hipStream_t stream, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t,
                          int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t, int32_t);
