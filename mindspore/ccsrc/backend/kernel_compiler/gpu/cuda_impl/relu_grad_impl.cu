#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "backend/kernel_compiler/gpu/cuda_impl/relu_grad_impl.cuh"
#include "runtime/device/gpu/cuda_common.h"

template <typename T>
__global__ void CalReLUGradKernel(int size, T *dy, T *y, T *dx) {
  for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    dx[pos] = y[pos] > static_cast<T>(0) ? dy[pos] : static_cast<T>(0);
  }
}

template <typename T>
void CalReLUGrad(int size, T *dy, T *y, T *dx, hipStream_t hip_stream) {
  CalReLUGradKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, dy, y, dx);
  return;
}

template <typename T>
__global__ void PReluChannelSharedGradKernel(size_t size, T *dy_addr, T *x_addr, T *w_addr, T *dx_addr, T *dwc_addr) {
  T zero = static_cast<T>(0);
  T w = w_addr[0];
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    T dy = dy_addr[pos];
    T x = x_addr[pos];
    dx_addr[pos] = x > zero ? dy : w * dy;
    dwc_addr[pos] = x > zero ? zero : x * dy;
  }
}

template <typename T>
void PReluChannelSharedGrad(size_t input_size, T *dy_addr, T *x_addr, T *w_addr, T *dx_addr, T *dwc_addr,
                            hipStream_t hip_stream) {
  PReluChannelSharedGradKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, hip_stream>>>(input_size, dy_addr, x_addr,
                                                                                        w_addr, dx_addr, dwc_addr);
  return;
}

template void CalReLUGrad(int size, double *dy, double *y, double *dx, hipStream_t hip_stream);
template void CalReLUGrad(int size, float *dy, float *y, float *dx, hipStream_t hip_stream);
template void CalReLUGrad(int size, half *dy, half *y, half *dx, hipStream_t hip_stream);
template void CalReLUGrad(int size, int8_t *dy, int8_t *y, int8_t *dx, hipStream_t hip_stream);
template void CalReLUGrad(int size, int16_t *dy, int16_t *y, int16_t *dx, hipStream_t hip_stream);
template void CalReLUGrad(int size, int32_t *dy, int32_t *y, int32_t *dx, hipStream_t hip_stream);
template void CalReLUGrad(int size, int64_t *dy, int64_t *y, int64_t *dx, hipStream_t hip_stream);
template void CalReLUGrad(int size, uint8_t *dy, uint8_t *y, uint8_t *dx, hipStream_t hip_stream);
template void PReluChannelSharedGrad(size_t input_size, float *dy_addr, float *x_addr, float *w_addr, float *dx_addr,
                                     float *dwc_addr, hipStream_t hip_stream);
template void PReluChannelSharedGrad(size_t input_size, half *dy_addr, half *x_addr, half *w_addr, half *dx_addr,
                                     half *dwc_addr, hipStream_t hip_stream);
