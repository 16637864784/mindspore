#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/non_zero_impl.cuh"
#include <hipcub/hipcub.hpp>
#include <algorithm>

template <typename DataType>
struct IsZero {
  __host__ __device__ __forceinline__ size_t operator()(const DataType &x) const { return x == DataType(0) ? 0 : 1; }
};

template <typename IndexType>
__global__ void NonZeroKernel(const size_t *index_ptr, const size_t *shape_ptr, IndexType *output_ptr,
                              size_t input_size, size_t rank) {
  for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < input_size; tid += blockDim.x * gridDim.x) {
    bool is_write = (tid != 0 && index_ptr[tid] != index_ptr[tid - 1]) || (tid == 0 && index_ptr[tid]);
    if (is_write) {
      size_t fill_index = index_ptr[tid] * rank - 1;
      size_t fill_value = tid;
      for (size_t i = 0; i < rank; i++) {
        size_t base = shape_ptr[rank - 1 - i];
        output_ptr[fill_index] = fill_value % base;
        fill_index--;
        fill_value /= base;
      }
    }
  }
}

template <typename DataType, typename IndexType>
CUDA_LIB_EXPORT void NonZero(const DataType *input_ptr, size_t *index_ptr, size_t *shape_ptr, IndexType *output_ptr,
                             size_t input_size, size_t rank, hipStream_t hip_stream) {
  hipcub::TransformInputIterator<size_t, IsZero<DataType>, const DataType *> iter(input_ptr, IsZero<DataType>());
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  (void)hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes, iter, index_ptr, input_size, hip_stream);
  (void)hipMalloc(&d_temp_storage, temp_storage_bytes);
  (void)hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, iter, index_ptr, input_size, hip_stream);

  // Extract the first index to appear and transform into output index,
  // e.g., [0, 0, 1, 2, 2, 2] -> [(1, 2), (2, 3)] -> [(0, 0, 2), (0, 1, 0)] when shape is (2, 1, 3)
  NonZeroKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, hip_stream>>>(index_ptr, shape_ptr, output_ptr, input_size,
                                                                         rank);
  // Since hipGetLastError can return the last error from a runtime call,
  // we catch the error in Launch function.
  (void)hipFree(d_temp_storage);
}

template CUDA_LIB_EXPORT void NonZero<bool, int64_t>(const bool *input_ptr, size_t *index_ptr, size_t *shape_ptr,
                                                     int64_t *output_ptr, size_t input_size, size_t rank,
                                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void NonZero<uint8_t, int64_t>(const uint8_t *input_ptr, size_t *index_ptr, size_t *shape_ptr,
                                                        int64_t *output_ptr, size_t input_size, size_t rank,
                                                        hipStream_t hip_stream);
template CUDA_LIB_EXPORT void NonZero<uint16_t, int64_t>(const uint16_t *input_ptr, size_t *index_ptr,
                                                         size_t *shape_ptr, int64_t *output_ptr, size_t input_size,
                                                         size_t rank, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void NonZero<uint32_t, int64_t>(const uint32_t *input_ptr, size_t *index_ptr,
                                                         size_t *shape_ptr, int64_t *output_ptr, size_t input_size,
                                                         size_t rank, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void NonZero<uint64_t, int64_t>(const uint64_t *input_ptr, size_t *index_ptr,
                                                         size_t *shape_ptr, int64_t *output_ptr, size_t input_size,
                                                         size_t rank, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void NonZero<int8_t, int64_t>(const int8_t *input_ptr, size_t *index_ptr, size_t *shape_ptr,
                                                       int64_t *output_ptr, size_t input_size, size_t rank,
                                                       hipStream_t hip_stream);
template CUDA_LIB_EXPORT void NonZero<int16_t, int64_t>(const int16_t *input_ptr, size_t *index_ptr, size_t *shape_ptr,
                                                        int64_t *output_ptr, size_t input_size, size_t rank,
                                                        hipStream_t hip_stream);
template CUDA_LIB_EXPORT void NonZero<int32_t, int64_t>(const int32_t *input_ptr, size_t *index_ptr, size_t *shape_ptr,
                                                        int64_t *output_ptr, size_t input_size, size_t rank,
                                                        hipStream_t hip_stream);
template CUDA_LIB_EXPORT void NonZero<int64_t, int64_t>(const int64_t *input_ptr, size_t *index_ptr, size_t *shape_ptr,
                                                        int64_t *output_ptr, size_t input_size, size_t rank,
                                                        hipStream_t hip_stream);
template CUDA_LIB_EXPORT void NonZero<half, int64_t>(const half *input_ptr, size_t *index_ptr, size_t *shape_ptr,
                                                     int64_t *output_ptr, size_t input_size, size_t rank,
                                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void NonZero<float, int64_t>(const float *input_ptr, size_t *index_ptr, size_t *shape_ptr,
                                                      int64_t *output_ptr, size_t input_size, size_t rank,
                                                      hipStream_t hip_stream);
template CUDA_LIB_EXPORT void NonZero<double, int64_t>(const double *input_ptr, size_t *index_ptr, size_t *shape_ptr,
                                                       int64_t *output_ptr, size_t input_size, size_t rank,
                                                       hipStream_t hip_stream);
