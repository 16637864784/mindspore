#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/selu_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/cuda_common.h"
#include "include/hip/hip_runtime.h"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void CalculateSeluKernel(const T *input, const size_t input_elements, T scale_dot_alpha, T scale,
                                    T *output) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < input_elements; i += blockDim.x * gridDim.x) {
    T input_value = input[i];
    T template_zero = static_cast<T>(0.0);
    output[i] = input_value >= template_zero ? scale * input_value : scale_dot_alpha * expm1(input_value);
  }
}

__global__ void CalculateSeluKernel(const half *input, const size_t input_elements, half scale_dot_alpha, half scale,
                                    half *output) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < input_elements; i += blockDim.x * gridDim.x) {
    half input_value = input[i];
    half template_zero = static_cast<half>(0.0);
    output[i] = input_value >= template_zero ? scale * input_value
                                             : scale_dot_alpha * static_cast<half>(expm1(__half2float(input_value)));
  }
}

template <typename T>
void CalculateSelu(const T *input, size_t input_elements, T scale_dot_alpha, T scale, T *output,
                   hipStream_t hip_stream) {
  CalculateSeluKernel<<<GET_BLOCKS(input_elements), GET_THREADS, 0, hip_stream>>>(input, input_elements,
                                                                                   scale_dot_alpha, scale, output);
}

template CUDA_LIB_EXPORT void CalculateSelu<double>(const double *input, size_t input_elements, double scale_dot_alpha,
                                                    double scale, double *output, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalculateSelu<float>(const float *input, size_t input_elements, float scale_dot_alpha,
                                                   float scale, float *output, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalculateSelu<half>(const half *input, size_t input_elements, half scale_dot_alpha,
                                                  half scale, half *output, hipStream_t hip_stream);
