#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include "include/hip/hip_fp16.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/apply_adagrad_d_a_impl.cuh"

template <typename T>
__device__ __forceinline__ T SqrtFunc(T input) {
  return sqrt(input);
}

template <>
__device__ __forceinline__ half SqrtFunc(half input) {
  return hsqrt(input);
}

template <typename T>
__device__ __forceinline__ T AbsFunc(T x) {
  return abs(x);
}

template <>
__device__ __forceinline__ half AbsFunc(half x) {
  return abs(__half2float(x));
}

template <typename T>
__device__ __forceinline__ T MaxFunc(T x, T y) {
  return max(x, y);
}

template <>
__device__ __forceinline__ half MaxFunc(half x, half y) {
  return max(__half2float(x), __half2float(y));
}

template <typename T>
__device__ __forceinline__ T Sign(T num) {
  if (num > static_cast<T>(0.0)) {
    return static_cast<T>(1.0);
  } else if (num == static_cast<T>(0.0)) {
    return static_cast<T>(0.0);
  } else {
    return static_cast<T>(-1.0);
  }
}

template <typename T, typename S>
__global__ void ApplyAdagradDAKernel(const size_t batch_size, const size_t size, T *var, T * accum, T *squared_accum,
                                     const T *grad, const T *lr, const T *l1, const T *l2, const S *global_step,
                                     T *output_var, T *output_accum, T *output_squared_accum) {
  T zero = static_cast<T>(0.0);
  T minus_one = static_cast<T>(-1);
  for (size_t b = 0; b < batch_size; b++) {
    for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += gridDim.x * blockDim.x) {
      output_accum[pos] = accum[pos] + grad[pos];
      output_squared_accum[pos] = squared_accum[pos] + grad[pos] * grad[pos];
      T tmp_val;
      if (lr[0] > zero) {
        T tmp_accum = AbsFunc(output_accum[pos]) - l1[0] * static_cast<T>(static_cast<double>(global_step[0]));
        tmp_val = Sign(output_accum[pos]) * MaxFunc(tmp_accum, zero);
      } else {
        tmp_val = output_accum[pos];
      }
      auto x_value = minus_one * lr[0] * tmp_val;
      auto y_value = l2[0] * static_cast<T>(static_cast<double>(global_step[0])) * lr[0] +
      SqrtFunc(output_squared_accum[pos]);
      output_var[pos] = x_value / y_value;
    }
    var = var + size;
    accum = accum + size;
    grad = grad + size;
  }
}

template <typename T, typename S>
void ApplyAdagradDA(const size_t batch_size, const size_t size, T *var, T * accum, T *squared_accum, const T *grad,
                    const T *lr, const T *l1, const T *l2, const S *global_step, T *output_var,
                    T *output_accum, T *output_squared_accum,
                    const uint32_t &device_id, hipStream_t hip_stream) {
  ApplyAdagradDAKernel<<<CUDA_BLOCKS(device_id, size), CUDA_THREADS(device_id), 0, hip_stream>>>(batch_size, size,
    var, accum, squared_accum, grad, lr, l1, l2, global_step, output_var, output_accum, output_squared_accum);
}

template CUDA_LIB_EXPORT void ApplyAdagradDA<float, int32_t>(const size_t batch_size, const size_t size, float *var,
                                                             float * accum, float *squared_accum, const float *grad,
                                                             const float *lr, const float *l1, const float *l2,
                                                             const int32_t *global_step, float *output_var,
                                                             float *output_accum, float *output_squared_accum,
                                                             const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ApplyAdagradDA<float, int64_t>(const size_t batch_size, const size_t size, float *var,
                                                             float * accum, float *squared_accum, const float *grad,
                                                             const float *lr, const float *l1, const float *l2,
                                                             const int64_t *global_step, float *output_var,
                                                             float *output_accum, float *output_squared_accum,
                                                             const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ApplyAdagradDA<half, int32_t>(const size_t batch_size, const size_t size, half *var,
                                                            half * accum, half *squared_accum, const half *grad,
                                                            const half *lr, const half *l1, const half *l2,
                                                            const int32_t *global_step, half *output_var,
                                                             half *output_accum, half *output_squared_accum,
                                                             const uint32_t &device_id, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ApplyAdagradDA<half, int64_t>(const size_t batch_size, const size_t size, half *var,
                                                            half * accum, half *squared_accum, const half *grad,
                                                            const half *lr, const half *l1, const half *l2,
                                                            const int64_t *global_step, half *output_var,
                                                            half *output_accum, half *output_squared_accum,
                                                            const uint32_t &device_id, hipStream_t hip_stream);
