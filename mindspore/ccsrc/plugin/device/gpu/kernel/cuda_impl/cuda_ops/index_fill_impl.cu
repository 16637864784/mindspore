#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/index_fill_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T>
__global__ void IndexFillKernel(T *out_ptr, const int *index_ptr, const size_t index_size, const size_t outer_size,
                                const int dim_size, const size_t inner_size, const T *value_ptr, bool *out_bound_ptr,
                                size_t stride1, size_t stride2) {
  for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < index_size; tid += blockDim.x * gridDim.x) {
    // Each index must be [-dim_size, dim_size)
    int index = index_ptr[tid / stride1];
    if (index < -dim_size || index >= dim_size) {
      *out_bound_ptr = true;
      break;
    } else if (index < 0) {
      index += dim_size;
    }
    size_t offset = tid % stride1;
    size_t inner_idx = offset % inner_size;
    size_t outer_idx = offset / inner_size;
    size_t out_idx = outer_idx * stride2 + index * inner_size + inner_idx;
    out_ptr[out_idx] = *value_ptr;
  }
}

template <typename T>
void IndexFill(T *out_ptr, const int *index_ptr, const size_t index_size, const size_t outer_size, const int dim_size,
               const size_t inner_size, const T *value_ptr, bool *out_bound_ptr, hipStream_t hip_stream) {
  size_t stride1 = outer_size * inner_size;
  size_t stride2 = dim_size * inner_size;
  IndexFillKernel<<<GET_BLOCKS(index_size), GET_THREADS, 0, hip_stream>>>(
    out_ptr, index_ptr, index_size, outer_size, dim_size, inner_size, value_ptr, out_bound_ptr, stride1, stride2);
}

template CUDA_LIB_EXPORT void IndexFill<double>(double *out_ptr, const int *index_ptr, const size_t index_size,
                                                const size_t outer_size, const int dim_size, const size_t inner_size,
                                                const double *value_ptr, bool *out_bound_ptr, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void IndexFill<float>(float *out_ptr, const int *index_ptr, const size_t index_size,
                                               const size_t outer_size, const int dim_size, const size_t inner_size,
                                               const float *value_ptr, bool *out_bound_ptr, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void IndexFill<half>(half *out_ptr, const int *index_ptr, const size_t index_size,
                                              const size_t outer_size, const int dim_size, const size_t inner_size,
                                              const half *value_ptr, bool *out_bound_ptr, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void IndexFill<int8_t>(int8_t *out_ptr, const int *index_ptr, const size_t index_size,
                                                const size_t outer_size, const int dim_size, const size_t inner_size,
                                                const int8_t *value_ptr, bool *out_bound_ptr, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void IndexFill<int16_t>(int16_t *out_ptr, const int *index_ptr, const size_t index_size,
                                                 const size_t outer_size, const int dim_size, const size_t inner_size,
                                                 const int16_t *value_ptr, bool *out_bound_ptr,
                                                 hipStream_t hip_stream);
template CUDA_LIB_EXPORT void IndexFill<int32_t>(int *out_ptr, const int *index_ptr, const size_t index_size,
                                                 const size_t outer_size, const int dim_size, const size_t inner_size,
                                                 const int32_t *value_ptr, bool *out_bound_ptr,
                                                 hipStream_t hip_stream);
template CUDA_LIB_EXPORT void IndexFill<int64_t>(int64_t *out_ptr, const int *index_ptr, const size_t index_size,
                                                 const size_t outer_size, const int dim_size, const size_t inner_size,
                                                 const int64_t *value_ptr, bool *out_bound_ptr,
                                                 hipStream_t hip_stream);
