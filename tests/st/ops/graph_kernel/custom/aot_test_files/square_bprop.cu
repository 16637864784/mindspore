
#include <hip/hip_runtime.h>
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#define THREADS 1024
__global__ void CustomSquareBpropKernel(float *input1, float *input3, float *output, size_t size) {
  auto idx = blockIdx.x * THREADS + threadIdx.x;
  if (idx < size) {
    output[idx] = input1[idx] * input3[idx] * 2;
  }
}

extern "C" int CustomSquareBprop(int nparam, void **params, int *ndims, int64_t **shapes, const char **dtypes,
                                 void *stream, void *extra) {
  hipStream_t custream = static_cast<hipStream_t>(stream);
  if (nparam != 4) return 1;
  void *input1 = params[0];
  void *input3 = params[2];
  void *output = params[3];

  size_t size = 1;

  for (int i = 0; i < ndims[3]; i++) {
    size *= shapes[3][i];
  }
  int n = size / THREADS;
  for (int i = 0; i < nparam; i++) {
    if (strcmp(dtypes[i], "float32") != 0) {
      return 2;
    }
  }

  CustomSquareBpropKernel<<<n + 1, THREADS, 0, custream>>>(static_cast<float *>(input1), static_cast<float *>(input3),
                                                           static_cast<float *>(output), size);
  return 0;
}
